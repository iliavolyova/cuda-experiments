#include "hip/hip_runtime.h"
#include	"cuda_auxiliary.h"

__device__ int x[128];

__global__ void kernel(void)
{
	x[threadIdx.x] += threadIdx.x;
}

int main()
{
	int	hst_x[128];
	int	*ptr;

	for (int i = 0; i < 128; ++i)
		hst_x[i] = i;

	hipMemcpyToSymbol(HIP_SYMBOL(x), hst_x, 128 * sizeof(int));
	
	kernel<<<1,128>>>();
	cuda_exec(hipDeviceSynchronize());

	// kopira u hst_x[0] peti element vektora x
	hipMemcpyFromSymbol(hst_x, HIP_SYMBOL(x), sizeof(int), 5 * sizeof(int));

	printf("%d\n\n", hst_x[0]);
	
	// u ptr dohvaćamo adresu na kojoj počinje polje x
	hipGetSymbolAddress((void **) &ptr, x);

	hipMemcpy(hst_x, ptr, 128 * sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < 128; ++i)
		printf("%d\n", hst_x[i]);

	return 0;
}
