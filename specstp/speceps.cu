#include "hip/hip_runtime.h"
#include	<stdio.h>
#include    "cuda_auxiliary.h"

#define BLOCK_SIZE 64

__global__ void gpu_dgemv(double *A, double *x, double *y, const int dim)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    double sum = 0;

    if (gid < dim){
        for (i = 0; i < dim; ++i){
            sum += A[(i*dim) + gid] * x[i];
        }
        y[gid] = sum;
    }

}

__global__ void gpu_dnrm2(double *x, double *nrm, const int dim)
{
    __shared__ double cache[BLOCK_SIZE];
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    if (gid < dim)
        cache[tid] = x[gid];
    else
        cache[tid] = 0;

    __syncthreads();

    cache[tid] = cache[tid] * cache[tid];

    __syncthreads();

    int i = blockDim.x / 2;
    while(i > 0){
        if(tid < i)
            cache[tid] = cache[tid] + cache[tid + i];
        __syncthreads();

        i >>= 1;
    }

    if (tid == 0) {
        nrm[0] = 1.0/sqrt(cache[0]);
    }
}

__global__ void gpu_dscal(double *x, double *y, double *alpha, const int dim)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < dim)
        y[gid] = x[gid] * alpha[0];
}

__global__ void gpu_subtract(double *x, double *y, double *out, const int dim)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < dim)
        out[gid] = x[gid] - y[gid];
}

__global__ void gpu_ddot(double *x, double *y, double *out, const int dim)
{
    __shared__ double cache[BLOCK_SIZE];
    int cacheindex = threadIdx.x;
    double temp;

    for(int gid = blockIdx.x * blockDim.x + threadIdx.x; gid < dim; gid += blockDim.x * gridDim.x)
        temp += x[gid] * y[gid];

    cache[cacheindex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i > 0) {
        if (cacheindex < i)
            cache[cacheindex] += cache[cacheindex + i];
        __syncthreads();

        i >>= 1;
    }

    if (threadIdx.x == 0)
        out[0] = cache[0];

}

int main(int argc, char **argv)
{
    FILE *fp_A = NULL;
    FILE *fp_x = NULL;

    double *hst_A = NULL;
    double *hst_x = NULL;

    double *dev_A = NULL;
    double *dev_x = NULL;
    double *dev_y = NULL;
    double *dev_tmp = NULL;
    double *dev_nrm_inv = NULL;
    double *dev_lambda;
    double *dev_subs;

    double eigval;
    double lambda;
    double subsnorm;
    double EPS = 0.00001;

    bool converged = false;

    dim3	block_size;
    dim3	grid_size;

    int dim;

    if (argc != 4) {
        fprintf(stderr, "usage: %s N A.dat x0.dat\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    dim = atoi(argv[1]);

    open_file(fp_A, argv[2], "r");
    host_alloc(hst_A, double, dim * dim);

    open_file(fp_x, argv[3], "r");
    host_alloc(hst_x, double, dim);

    read_file(hst_A, sizeof(double), dim * dim, fp_A);
    read_file(hst_x, sizeof(double), dim, fp_x);

    cuda_exec(hipMalloc(&dev_A, dim * dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_x, dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_y, dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_tmp, dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_subs, dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_nrm_inv, sizeof(double)));
    cuda_exec(hipMalloc(&dev_lambda, sizeof(double)));

    cuda_exec(hipMemcpy(dev_A, hst_A, dim * dim * sizeof(double), hipMemcpyHostToDevice));
    cuda_exec(hipMemcpy(dev_x, hst_x, dim * sizeof(double), hipMemcpyHostToDevice));
    cuda_exec(hipMemcpy(dev_y, hst_x, dim * sizeof(double), hipMemcpyHostToDevice));

    block_size.x = BLOCK_SIZE;
    grid_size.x = min((dim + block_size.x - 1) / block_size.x, 65535);

    int cnt = 0;
    while(!converged){
        gpu_dnrm2<<<grid_size, block_size>>>(dev_y, dev_nrm_inv, dim);
        gpu_dscal<<<grid_size, block_size>>>(dev_y, dev_x, dev_nrm_inv, dim);
        gpu_dgemv<<<grid_size, block_size>>>(dev_A, dev_x, dev_y, dim);
        gpu_ddot<<<grid_size, block_size>>>(dev_x, dev_y, dev_lambda, dim);

        gpu_dscal<<<grid_size, block_size>>>(dev_x, dev_x, dev_lambda, dim);
        gpu_subtract<<<grid_size, block_size>>>(dev_y, dev_x, dev_tmp, dim);
        gpu_dnrm2<<<grid_size, block_size>>>(dev_tmp, dev_nrm_inv, dim);

        cuda_exec(hipMemcpy(&lambda, dev_lambda, sizeof(double), hipMemcpyDeviceToHost));
        cuda_exec(hipMemcpy(&subsnorm, dev_nrm_inv, sizeof(double), hipMemcpyDeviceToHost));

        printf(" %#.16lg\t %#.16lg\n", subsnorm, lambda * EPS);
        if (subsnorm < EPS * lambda || cnt == 10)
            converged = true;
        cnt++;
    }

    cuda_exec(hipMemcpy(&eigval, dev_lambda, sizeof(double), hipMemcpyDeviceToHost));
    printf("\nSpectrum: %#.16lg\n", eigval);

    hipFree(dev_A);
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_tmp);
    hipFree(dev_subs);
    hipFree(dev_nrm_inv);
    hipFree(dev_lambda);

    host_free(hst_A);
    host_free(hst_x);

    return 0;
}

