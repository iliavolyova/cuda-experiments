#include "hip/hip_runtime.h"
#include	<stdio.h>
#include    "cuda_auxiliary.h"

#define BLOCK_SIZE 64

__global__ void gpu_dgemv(double *A, double *x, double *y, const int dim)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int i;
    double sum = 0;

    if (gid < dim){
        for (i = 0; i < dim; ++i){
            sum += A[(i*dim) + gid] * x[i];
        }
        y[gid] = sum;
    }

}

__global__ void gpu_dnrm2(double *x, double *nrm, const int dim)
{
    __shared__ double cache[BLOCK_SIZE];
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    if (gid < dim)
        cache[tid] = x[gid];
    else
        cache[tid] = 0;

    __syncthreads();

    cache[tid] = cache[tid] * cache[tid];

    __syncthreads();

    int i = blockDim.x / 2;
    while(i > 0){
        if(tid < i)
            cache[tid] = cache[tid] + cache[tid + i];
        __syncthreads();

        i >>= 1;
    }

    if (tid == 0) {
        nrm[0] = sqrt(cache[0]);
    }
}

__global__ void gpu_dscal(double *x, double alpha, const int n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    x[gid] *= alpha;
}

__global__ void gpu_ddot(double *x, double *y, double *out, const int dim)
{
    __shared__ double cache[BLOCK_SIZE];
    int cacheindex = threadIdx.x;
    double temp;

    for(int gid = blockIdx.x * blockDim.x + threadIdx.x; gid < dim; gid += blockDim.x * gridDim.x)
        temp += x[gid] * y[gid];

    cache[cacheindex] = temp;

    __syncthreads();

    int i = blockDim.x / 2;
    while (i > 0) {
        if (cacheindex < i)
            cache[cacheindex] += cache[cacheindex + i];
        __syncthreads();

        i >>= 1;
    }

    if (threadIdx.x = 0)
        out[0] = cache[0];

}

int main(int argc, char **argv)
{
    FILE *fp_A = NULL;
    FILE *fp_x = NULL;

    double *hst_A = NULL;
    double *hst_x = NULL;

    double *dev_A = NULL;
    double *dev_x = NULL;
    double *dev_y = NULL;
    double *dev_nrm = NULL;

    double norm;
    double eigval;

    const double ONE = 1.0;
    const double ZERO = 0.0;
    double alpha;
    bool converged = false;

    dim3	block_size;
    dim3	grid_size;

    int dim;

    if (argc != 4) {
        fprintf(stderr, "usage: %s N A.dat x0.dat\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    dim = atoi(argv[1]);

    open_file(fp_A, argv[2], "r");
    host_alloc(hst_A, double, dim * dim);

    open_file(fp_x, argv[3], "r");
    host_alloc(hst_x, double, dim);

    read_file(hst_A, sizeof(double), dim * dim, fp_A);
    read_file(hst_x, sizeof(double), dim, fp_x);

    cuda_exec(hipMalloc(&dev_A, dim * dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_x, dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_y, dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_nrm, sizeof(double)));

    cuda_exec(hipMemcpy(dev_A, hst_A, dim * dim * sizeof(double), hipMemcpyHostToDevice));
    cuda_exec(hipMemcpy(dev_x, hst_x, dim * sizeof(double), hipMemcpyHostToDevice));
    cuda_exec(hipMemcpy(dev_y, hst_x, dim * sizeof(double), hipMemcpyHostToDevice));

    block_size.x = BLOCK_SIZE;
    grid_size.x = min((dim + block_size.x - 1) / block_size.x, 65535);

    int i;
    while(!converged){
        gpu_dnrm2<<grid_size, block_size>>(dev_y, dev_nrm, dim);


        converged = true;
    }

    cuda_exec(hipMemcpy(&eigval, dev_nrm, sizeof(double), hipMemcpyDeviceToHost));


    printf("\nSpectrum: %#.16lg\n", eigval);

    hipFree(dev_A);
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_nrm);

    host_free(hst_A);
    host_free(hst_x);

    return 0;
}

