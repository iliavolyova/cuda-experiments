#include "hip/hip_runtime.h"
#include	<stdio.h>
#include    "cuda_auxiliary.h"

/*
 * compile: nvcc specstp.cu -lcublas -o specstp
 */

int main(int argc, char **argv)
{
    hipblasHandle_t cublas_handle;

    FILE *fp_A = NULL;
    FILE *fp_x = NULL;

    double *hst_A = NULL;
    double *hst_x = NULL;

    double *dev_A = NULL;
    double *dev_x = NULL;
    double *dev_y = NULL;

    double norm;
    double eigval;

    const double ONE = 1.0;
    const double ZERO = 0.0;
    double alpha;

    int dim;
    int steps;

    if (argc != 5) {
        fprintf(stderr, "usage: %s N A.dat x0.dat steps\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    dim = atoi(argv[1]);
    steps = atoi(argv[4]);

    open_file(fp_A, argv[2], "r");
    host_alloc(hst_A, double, dim * dim);

    open_file(fp_x, argv[3], "r");
    host_alloc(hst_x, double, dim);

    read_file(hst_A, sizeof(double), dim * dim, fp_A);
    read_file(hst_x, sizeof(double), dim, fp_x);

    cuda_exec(hipMalloc(&dev_A, dim * dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_x, dim * sizeof(double)));
    cuda_exec(hipMalloc(&dev_y, dim * sizeof(double)));

    cublas_exec(hipblasCreate(&cublas_handle));
    cublas_exec(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST));

    cublas_exec(hipblasSetMatrix(dim, dim, sizeof(double), hst_A, dim, dev_A, dim));
    cublas_exec(hipblasSetVector(dim, sizeof(double), hst_x, 1, dev_x, 1));
    cublas_exec(hipblasSetVector(dim, sizeof(double), hst_x, 1, dev_y, 1));

    int i;
    for (i = 0; i < steps; ++i){
        cublas_exec(hipblasDnrm2(cublas_handle, dim, dev_y, 1, &norm));
        alpha = 1.0/norm;
        cublas_exec(hipblasDscal(cublas_handle, dim, &alpha, dev_y, 1));
        cublas_exec(hipblasDcopy(cublas_handle, dim, dev_y, 1, dev_x, 1));

        cublas_exec(hipblasDgemv(cublas_handle, HIPBLAS_OP_T, dim, dim, &ONE, dev_A, dim, dev_x, 1, &ZERO, dev_y, 1));
    }

    cublas_exec(hipblasDdot(cublas_handle, dim, dev_x, 1, dev_y, 1, &eigval));
    printf("\nSpectrum: %#.16lg\n", eigval);

    cublas_exec(hipblasDestroy(cublas_handle));
    hipFree(dev_A);
    hipFree(dev_x);
    hipFree(dev_y);

    host_free(hst_A);
    host_free(hst_x);

    return 0;
}

